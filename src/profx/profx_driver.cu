#include "hip/hip_runtime.h"
// ==============================================================================
// This file is part of THOR.
//
//     THOR is free software : you can redistribute it and / or modify
//     it under the terms of the GNU General Public License as published by
//     the Free Software Foundation, either version 3 of the License, or
//     (at your option) any later version.
//
//     THOR is distributed in the hope that it will be useful,
//     but WITHOUT ANY WARRANTY; without even the implied warranty of
//     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
//     GNU General Public License for more details.
//
//     You find a copy of the GNU General Public License in the main
//     THOR directory under <license.txt>.If not, see
//     <http://www.gnu.org/licenses/>.
// ==============================================================================
//
//
//
//
// Description: Physics modules.
//
//
// Method: This version just includes the held-suarez test.
//
// Known limitations: None
//
// Known issues: None
//
// If you use this code please cite the following reference:
//
//       [1] Mendonca, J.M., Grimm, S.L., Grosheintz, L., & Heng, K., ApJ, 829, 115, 2016
//
// Current Code Owner: Joao Mendonca, EEG. joao.mendonca@csh.unibe.ch
//
// History:
// Version Date       Comment
// ======= ====       =======
//
// 1.0     16/08/2017 Released version  (JM)
//
////////////////////////////////////////////////////////////////////////

#include "../headers/esp.h"
#include "../headers/phy/profx_auxiliary.h"
#include "../headers/phy/profx_held_suarez.h"
#include "../headers/phy/profx_shallowHJ_hs.h"
#include "../headers/phy/profx_deepHJ_hs.h"
#include "../headers/phy/profx_tidalearth_hs.h"
#include "../headers/phy/profx_H_recomb.h"
#include "../headers/phy/apocalypse_sponge.h"
#include "../headers/phy/profx_RT.h"

#include "binary_test.h"
#include "debug_helpers.h"


__host__ void ESP::ProfX(int    planetnumber, // Planet ID
                         int    nstep       , // Step number
                         int    hstest      , // Held-Suarez test option
                         double time_step   , // Time-step [s]
                         double Omega       , // Rotation rate [1/s]
                         double Cp          , // Specific heat capacity [J/kg/K]
                         double Rd          , // Gas constant [J/kg/K]
                         double Mmol        , // Mean molecular mass of dry air [kg]
                         double mu          , // Atomic mass unit [kg]
                         double kb          , // Boltzmann constant [J/K]
                         double P_Ref       , // Reference pressure [Pa]
                         double Gravit      , // Gravity [m/s^2]
                         double A           ,// Planet radius [m]
<<<<<<< HEAD
                         bool   sponge      ,// sponge layer on/off
                         bool hh2recomb     ){// option of atomic H<->H2
=======
                         bool   sponge      , // Use sponge layer?
                         bool   shrink_sponge){ // Shrink sponge after some time
>>>>>>> master
    USE_BENCHMARK()
//
//  Number of threads per block.
    const int NTH = 256;

//  Specify the block sizes.
    dim3 NB((point_num / NTH) + 1, nv, 1);
    dim3 NBRT((point_num/NTH) + 1, 1, 1);

    if (sponge==true) {
      dim3 NBT((point_num / NTH) + 1, nv, 1);

      hipMemset(vbar_d, 0, sizeof(double) * 3 * nlat * nv);
      zonal_v <<<NBT,NTH >>>(Mh_d                     ,
                             W_d                       ,
                             Rho_d                    ,
                             vbar_d                    ,
                             zonal_mean_tab_d,
                             lonlat_d                  ,
                             point_num                        );

      hipDeviceSynchronize();

      if (shrink_sponge == true) {
        if (nstep*time_step >= t_shrink*86400) {
          ns_sponge = 1.0 - 0.5*(1.0-ns_sponge);
          shrink_sponge = false;
        }
      }

      sponge_layer <<< NB,NTH >>>(Mh_d                      ,
                                  Rho_d                    ,
                                  W_d                       ,
                                  Wh_d                     ,
                                  vbar_d                    ,
                                  zonal_mean_tab_d,
                                  lonlat_d                  ,
                                  Altitude_d               ,
                                  Altitudeh_d             ,
                                  Rv_sponge             ,
                                  ns_sponge              ,
                                  time_step                  ,
                                  nlat                       ,
                                  point_num                  ,
                                  nv                           );
    }

//  Computes the initial temperature.
    Compute_temperature <<< NB, NTH >>> (temperature_d,
                                         pt_d         ,
                                         pressure_d   ,
                                         Rho_d        ,
                                         P_Ref        ,
                                         Rd           ,
                                         Cp           ,
                                         point_num    );

    BENCH_POINT_I(current_step, "phy_T", vector<string>({}), vector<string>({"Rho_d", "pressure_d", "Mh_d", "Wh_d", "temperature_d", "W_d"}))
//  Check for nan.
    check_h = check_array_for_nan(temperature_d,nv*point_num,1,check_d);
    if(check_h){
       printf("\n\n Error in NAN check after PROFX:compute_temp!\n");
       exit(EXIT_FAILURE);
    }

///////////////////////
// HELD SUAREZ TEST  //
///////////////////////
//
    if (planetnumber == 1) {
      if (hstest == 1) {
        hipDeviceSynchronize();
        held_suarez<<< NB, NTH >>> (Mh_d         ,
                                    pressure_d   ,
                                    Rho_d        ,
                                    temperature_d,
                                    Gravit       ,
                                    Cp           ,
                                    Rd           ,
                                    Altitude_d   ,
                                    Altitudeh_d  ,
                                    lonlat_d     ,
                                    time_step    ,
                                    point_num    );
      } else if (hstest == 2) {
        hipDeviceSynchronize();
        tidalearth_hs<<< NB, NTH >>> (Mh_d         ,
                                    pressure_d   ,
                                    Rho_d        ,
                                    temperature_d,
                                    Gravit       ,
                                    Cp           ,
                                    Rd           ,
                                    Altitude_d   ,
                                    Altitudeh_d  ,
                                    lonlat_d     ,
                                    time_step    ,
                                    point_num    );
      } else if (hstest == 3) {
        hipDeviceSynchronize();
        shallowHJ_hs<<< NB, NTH >>> (Mh_d         ,
                                    pressure_d   ,
                                    Rho_d        ,
                                    temperature_d,
                                    Gravit       ,
                                    Cp           ,
                                    Rd           ,
                                    Altitude_d   ,
                                    Altitudeh_d  ,
                                    lonlat_d     ,
                                    time_step    ,
                                    point_num    );
      } else if (hstest == 4) {
        hipDeviceSynchronize();
        deepHJ_hs<<< NB, NTH >>> (Mh_d         ,
                                    pressure_d   ,
                                    Rho_d        ,
                                    temperature_d,
                                    Gravit       ,
                                    Cp           ,
                                    Rd           ,
                                    Altitude_d   ,
                                    Altitudeh_d  ,
                                    lonlat_d     ,
                                    time_step    ,
                                    point_num    );
      }
      if (hh2recomb) {
        hipDeviceSynchronize();
        recomb_H<<< NB, NTH >>> (Mh_d         ,
                                    pressure_d   ,
                                    Rho_d        ,
                                    temperature_d,
                                    mixH_d       ,
                                    Gravit       ,
                                    Cp           ,
                                    Rd           ,
                                    Altitude_d   ,
                                    Altitudeh_d  ,
                                    lonlat_d     ,
                                    time_step    ,
                                    point_num    );
        hipDeviceSynchronize();
        ComputeMixH<<< NB, NTH >>> (temperature_d,
                                    pt_d         ,
                                    pressure_d   ,
                                    Rho_d        ,
                                    mixH_d       ,
                                    P_Ref        ,
                                    Rd           ,
                                    Cp           ,
                                    point_num          );
      }
    }
//
////////////////////////

    if(planetnumber != 1){
        printf("Planet value incorrect! (see in file planet.h)");
        exit(EXIT_FAILURE);
    }

    if (!hstest) {
        hipDeviceSynchronize();
        rtm_dual_band <<< NBRT, NTH >>> (pressure_d   ,
      //rtm_dual_band <<< 1,1 >>> (pressure_d         ,
                                       Rho_d        ,
                                       temperature_d,
                                       fnet_up_d    ,
                                       fnet_dn_d    ,
                                       tau_d        ,
                                       Gravit       ,
                                       Cp           ,
                                       lonlat_d     ,
                                       Altitude_d   ,
                                       Altitudeh_d  ,
                                       phtemp       ,
                                       dtemp        ,
                                       ttemp        ,
                                       thtemp       ,
                                       time_step    ,
                                       Tstar        ,
                                       planet_star_dist,
                                       radius_star  ,
                                       diff_fac     ,
                                       Tlow         ,
                                       albedo       ,
                                       tausw        ,
                                       taulw        ,
                                       incflx       ,
                                       P_Ref        ,
                                       point_num    ,
                                       nv           ,
                                       nvi          ,
                                       A             );
    }
    check_h = false;
    hipMemcpy(check_d, &check_h, sizeof(bool), hipMemcpyHostToDevice);
    isnan_check<<< 16, NTH >>>(temperature_d, nv, point_num, check_d);
    hipMemcpy(&check_h, check_d, sizeof(bool), hipMemcpyDeviceToHost);
    if(check_h){
       printf("\n\n Error in NAN check after PROFX:RT!\n");
       exit(EXIT_FAILURE);
    }

    BENCH_POINT_I(current_step, "phy_hstest", vector<string>({}), vector<string>({"Rho_d", "pressure_d", "Mh_d", "Wh_d", "temperature_d", "W_d"}))
//  Computes the new pressures.
    hipDeviceSynchronize();
    Compute_pressure <<< NB, NTH >>> (pressure_d   ,
                                      temperature_d,
                                      Rho_d        ,
                                      Rd           ,
                                      point_num    );

    check_h = false;
    hipMemcpy(check_d, &check_h, sizeof(bool), hipMemcpyHostToDevice);
    isnan_check<<< 16, NTH >>>(temperature_d, nv, point_num, check_d);
    hipMemcpy(&check_h, check_d, sizeof(bool), hipMemcpyDeviceToHost);
    if(check_h){
       printf("\n\n Error in NAN check after PROFX:compute_pressure!\n");
       exit(EXIT_FAILURE);
    }

#ifdef BENCHMARKING
    // recompute temperature from pressure and density, to avoid rounding issues when comparing
    Compute_temperature_only <<< NB, NTH >>> (temperature_d,
                                              pressure_d   ,
                                              Rho_d        ,
                                              Rd           ,
                                              point_num    );
#endif // BENCHMARKING

    BENCH_POINT_I(current_step, "phy_END", vector<string>({}), vector<string>({"Rho_d", "pressure_d", "Mh_d", "Wh_d", "temperature_d", "W_d"}))
//
//END OF INTEGRATION
//
}
