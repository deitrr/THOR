#include "hip/hip_runtime.h"
// ==============================================================================
// This file is part of THOR.
//
//     THOR is free software : you can redistribute it and / or modify
//     it under the terms of the GNU General Public License as published by
//     the Free Software Foundation, either version 3 of the License, or
//     (at your option) any later version.
//
//     THOR is distributed in the hope that it will be useful,
//     but WITHOUT ANY WARRANTY; without even the implied warranty of
//     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
//     GNU General Public License for more details.
//
//     You find a copy of the GNU General Public License in the main
//     THOR directory under <license.txt>.If not, see
//     <http://www.gnu.org/licenses/>.
// ==============================================================================
//
//
//
//
// Description: DYNAMICAL CORE INTEGRATION
//
// Method: RK3 Method & Forward-Backward Method
//
// Known limitations:
//   - It does not include a shock capture scheme.
//
// Known issues:
//   - Operational in just one GPU.
//
// If you use this code please cite the following reference:
//
//       [1] Mendonca, J.M., Grimm, S.L., Grosheintz, L., & Heng, K., ApJ, 829, 115, 2016
//
// Current Code Owners: Joao Mendonca (joao.mendonca@space.dtu.dk)
//                      Russell Deitrick (russell.deitrick@csh.unibe.ch)
//                      Urs Schroffenegger (urs.schroffenegger@csh.unibe.ch)
//
// History:
// Version Date       Comment
// ======= ====       =======
// 2.0     30/11/2018 Released version (RD & US)
// 1.0     16/08/2017 Released version  (JM)
//
////////////////////////////////////////////////////////////////////////

#include "esp.h" // Global parameters.

#include "dyn/thor_adv_cor.h"      // Advection term.
#include "dyn/thor_auxiliary.h"    // Temperature, interal energy, potential tempareture and effective gravity.
#include "dyn/thor_diff.h"         // Hyper-diffusion.
#include "dyn/thor_div.h"          // Divergence damping.
#include "dyn/thor_fastmodes.h"    // Fast terms.
#include "dyn/thor_slowmodes.h"    // Slow terms.
#include "dyn/thor_vertical_int.h" // Vertical momentum.

#include "binary_test.h"
#include "debug_helpers.h"

#include "phy_modules.h"

__host__ void ESP::Thor(const SimulationSetup& sim) {
    const int NTH = 256;

    // Vertical Eq only works on vertical stack of data, can run independently, only uses shared
    // memory for intermediate data that is not shared with neighbours.
    // Need to set the block size so that the internal arrays fit in shared memory for each block
    const int num_th_vertical_eq = 32;
    //  Specify the block sizes.
    const int LN = 16;                     // Size of the inner region side.
    dim3      NT(nl_region, nl_region, 1); // Number of threads in a block.
    dim3      NB(nr, nv, 1);               // Number of blocks.
    dim3      NBD(nr, nv, 6);              // Number of blocks in the diffusion routine.
    dim3      NBDP(2, nv, 6);              // Number of blocks in the diffusion routine. (POLES)
    dim3      NBP(2, nv, 1);               // Number of blocks. (POLES)


    //  Number of Small steps
    double ns_totald = 6; // Maximum number of small steps in a large step (double ).
    int    ns_totali = 6; // Maximum number of small steps in a large step (integer).
    int    ns_it;         // Number of small steps in each large step.
    double times;         // Sub-timestep.

    //  Initialize local variables used for the time integration.
    hipDeviceSynchronize();
    hipMemcpy(Mhk_d, Mh_d, point_num * nv * 3 * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(Whk_d, Wh_d, point_num * nvi * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(Wk_d, W_d, point_num * nv * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(Rhok_d, Rho_d, point_num * nv * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(pressurek_d, pressure_d, point_num * nv * sizeof(double), hipMemcpyDeviceToDevice);

    hipMemset(Mhs_d, 0, sizeof(double) * 3 * point_num * nv);
    hipMemset(Rhos_d, 0, sizeof(double) * point_num * nv);
    hipMemset(Whs_d, 0, sizeof(double) * point_num * nvi);
    hipMemset(Ws_d, 0, sizeof(double) * point_num * nv);
    hipMemset(pressures_d, 0, sizeof(double) * point_num * nv);

    if (phy_modules_execute)
        phy_modules_dyn_core_loop_init(*this);

    USE_BENCHMARK();
    BENCH_POINT_I(current_step, "thor_init", (), ("Rho_d", "pressure_d", "Mh_d", "Wh_d", "temperature_d", "W_d" /*, "tracer_d", "tracers_d", "tracerk_d"*/));


    //  Loop for large time integration.
    for (int rk = 0; rk < 3; rk++) {
        //      Local variables to define the length (times) and the number of the small steps (ns_it).
        if (rk == 0) ns_it = 1;
        if (rk == 1) ns_it = ns_totali / 2;
        if (rk == 2) ns_it = ns_totali;

        if (rk == 0) times = timestep / 3.0;
        if (rk == 1) times = timestep / ns_totald;
        if (rk == 2) times = timestep / ns_totald;

        // initialise some memory

        //
        //      Compute advection and coriolis terms.
        hipMemset(Adv_d, 0, sizeof(double) * 3 * point_num * nv); // Sets every value of Adv_d to
                                                                   // zero.
        hipDeviceSynchronize();


        // Updates: Adv_d, v_d

        Compute_Advec_Cori1<LN, LN><<<NB, NT>>>((double3*)Adv_d,
                                                (double3*)v_d,
                                                (double3*)Mhk_d,
                                                (double3*)div_d,
                                                Wk_d,
                                                Rhok_d,
                                                Altitude_d,
                                                sim.A,
                                                (double3*)func_r_d,
                                                maps_d,
                                                nl_region,
                                                sim.DeepModel);
        // Updates: Adv_d, v_d
        Compute_Advec_Cori_Poles<6><<<2, 1>>>(Adv_d,
                                              v_d,
                                              Mhk_d,
                                              div_d,
                                              Wk_d,
                                              Rhok_d,
                                              Altitude_d,
                                              sim.A,
                                              func_r_d,
                                              point_local_d,
                                              point_num,
                                              nv,
                                              sim.DeepModel);

        hipDeviceSynchronize();
        // Updates: Adv_d
        Compute_Advec_Cori2<<<(point_num / NTH) + 1, NTH>>>(Adv_d,
                                                            v_d,
                                                            Whk_d,
                                                            Rhok_d,
                                                            Altitude_d,
                                                            Altitudeh_d,
                                                            sim.Omega,
                                                            sim.A,
                                                            nv,
                                                            point_num,
                                                            sim.DeepModel);

        //
        //      Computes temperature, internal energy, potential temperature and effective gravity.
        hipDeviceSynchronize();

        BENCH_POINT_I_S(current_step, rk, "Compute_Advec_Cori", (), ("Rho_d", "pressure_d", "Mh_d", "Wh_d", "temperature_d", "W_d", "Adv_d", "v_d"))

        // Updates: temperature_d, h_d, hh_d, pt_d, pth_d, gtil_d, gtilh_d
        Compute_Temperature_H_Pt_Geff<<<(point_num / NTH) + 1, NTH>>>(temperature_d,
                                                                      pressurek_d,
                                                                      Rhok_d,
                                                                      h_d,
                                                                      hh_d,
                                                                      pt_d,
                                                                      pth_d,
                                                                      gtil_d,
                                                                      gtilh_d,
                                                                      Whk_d,
                                                                      sim.P_Ref,
                                                                      sim.Gravit,
                                                                      sim.Cp,
                                                                      sim.Rd,
                                                                      Altitude_d,
                                                                      Altitudeh_d,
                                                                      point_num,
                                                                      nv);


        //      Initializes slow terms.
        hipDeviceSynchronize();

        BENCH_POINT_I_S(current_step, rk, "Compute_Temperature_H_Pt_Geff", (), ("temperature_d", "h_d", "hh_d", "pt_d", "pth_d", "gtil_d", "gtilh_d"))

        hipMemset(SlowMh_d, 0, sizeof(double) * 3 * point_num * nv);
        hipMemset(SlowWh_d, 0, sizeof(double) * point_num * nvi);
        hipMemset(SlowRho_d, 0, sizeof(double) * point_num * nv);
        hipMemset(Slowpressure_d, 0, sizeof(double) * point_num * nv);
        //
        //      Hyper-Diffusion.
        if (sim.HyDiff) {
            hipMemset(diff_d, 0, sizeof(double) * 6 * point_num * nv);
            hipDeviceSynchronize();
            //Updates: diffmh_d, diffw_d, diffrh_d, diffpr_d, diff_d
            Diffusion_Op<LN, LN><<<NBD, NT>>>(diffmh_d,
                                              diffw_d,
                                              diffrh_d,
                                              diffpr_d,
                                              diff_d,
                                              Mhk_d,
                                              Rhok_d,
                                              temperature_d,
                                              Wk_d,
                                              areasTr_d,
                                              nvecoa_d,
                                              nvecti_d,
                                              nvecte_d,
                                              func_r_d,
                                              Kdh4_d,
                                              Altitude_d,
                                              sim.A,
                                              sim.Rd,
                                              maps_d,
                                              nl_region,
                                              0,
                                              sim.DeepModel);
            //Updates: diffmh_d, diffw_d, diffrh_d, diffpr_d, diff_d
            Diffusion_Op_Poles<5><<<NBDP, 1>>>(diffmh_d,
                                               diffw_d,
                                               diffrh_d,
                                               diffpr_d,
                                               diff_d,
                                               Mhk_d,
                                               Rhok_d,
                                               temperature_d,
                                               Wk_d,
                                               func_r_d,
                                               areasTr_d,
                                               nvecoa_d,
                                               nvecti_d,
                                               nvecte_d,
                                               Kdh4_d,
                                               Altitude_d,
                                               Altitudeh_d,
                                               sim.A,
                                               sim.Rd,
                                               point_local_d,
                                               point_num,
                                               0,
                                               sim.DeepModel);
            hipDeviceSynchronize();
            //Updates: diffmh_d, diffw_d, diffrh_d, diffpr_d, diff_d
            Diffusion_Op<LN, LN><<<NBD, NT>>>(diffmh_d,
                                              diffw_d,
                                              diffrh_d,
                                              diffpr_d,
                                              diff_d,
                                              Mhk_d,
                                              Rhok_d,
                                              temperature_d,
                                              Wk_d,
                                              areasTr_d,
                                              nvecoa_d,
                                              nvecti_d,
                                              nvecte_d,
                                              func_r_d,
                                              Kdh4_d,
                                              Altitude_d,
                                              sim.A,
                                              sim.Rd,
                                              maps_d,
                                              nl_region,
                                              1,
                                              sim.DeepModel);
            //Updates: diffmh_d, diffw_d, diffrh_d, diffpr_d, diff_d
            Diffusion_Op_Poles<5><<<NBDP, 1>>>(diffmh_d,
                                               diffw_d,
                                               diffrh_d,
                                               diffpr_d,
                                               diff_d,
                                               Mhk_d,
                                               Rhok_d,
                                               temperature_d,
                                               Wk_d,
                                               func_r_d,
                                               areasTr_d,
                                               nvecoa_d,
                                               nvecti_d,
                                               nvecte_d,
                                               Kdh4_d,
                                               Altitude_d,
                                               Altitudeh_d,
                                               sim.A,
                                               sim.Rd,
                                               point_local_d,
                                               point_num,
                                               1,
                                               sim.DeepModel);

            BENCH_POINT_I_S_PHY(current_step, rk, "Diffusion_Op_Poles", (), ("diffmh_d", "diffw_d", "diffrh_d", "diffpr_d", "diff_d", "difftr_d"))
        }

        if (phy_modules_execute)
            phy_modules_dyn_core_loop_slow_modes(*this,
                                                 sim,
                                                 current_step,
                                                 times);

        BENCH_POINT_I_S_PHY(current_step, rk, "DivDamp", (), ("Rhos_d", "Rhok_d", "Mhs_d", "Mhk_d", "Whs_d", "Whk_d", "pressures_d", "pressurek_d", "pressure_d"))
        //
        //      Divergence damping
        hipMemset(DivM_d, 0, sizeof(double) * point_num * 3 * nv);
        hipMemset(divg_Mh_d, 0, sizeof(double) * point_num * 3 * nv);
        if (sim.DivDampP) {
            hipDeviceSynchronize();
            // Updates: DivM_d, divg_Mh_d
            DivM_Op<LN, LN><<<NB, NT>>>(DivM_d,
                                        divg_Mh_d,
                                        Mhk_d,
                                        Whk_d,
                                        Kdhz_d,
                                        areasTr_d,
                                        nvecoa_d,
                                        nvecti_d,
                                        nvecte_d,
                                        func_r_d,
                                        Altitudeh_d,
                                        Altitude_d,
                                        sim.A,
                                        maps_d,
                                        nl_region,
                                        0,
                                        sim.DeepModel);
            // Updates: DivM_d, divg_Mh_d
            DivM_Op_Poles<5><<<NBP, 1>>>(DivM_d,
                                         divg_Mh_d,
                                         Mhk_d,
                                         Whk_d,
                                         Kdhz_d,
                                         areasTr_d,
                                         nvecoa_d,
                                         nvecti_d,
                                         nvecte_d,
                                         func_r_d,
                                         Altitudeh_d,
                                         Altitude_d,
                                         sim.A,
                                         point_local_d,
                                         point_num,
                                         0,
                                         sim.DeepModel);

            hipDeviceSynchronize();
            // Updates: DivM_d, divg_Mh_d
            DivM_Op<LN, LN><<<NB, NT>>>(DivM_d,
                                        divg_Mh_d,
                                        Mhk_d,
                                        Whk_d,
                                        Kdhz_d,
                                        areasTr_d,
                                        nvecoa_d,
                                        nvecti_d,
                                        nvecte_d,
                                        func_r_d,
                                        Altitudeh_d,
                                        Altitude_d,
                                        sim.A,
                                        maps_d,
                                        nl_region,
                                        1,
                                        sim.DeepModel);
            // Updates: DivM_d, divg_Mh_d
            DivM_Op_Poles<5><<<NBP, 1>>>(DivM_d,
                                         divg_Mh_d,
                                         Mhk_d,
                                         Whk_d,
                                         Kdhz_d,
                                         areasTr_d,
                                         nvecoa_d,
                                         nvecti_d,
                                         nvecte_d,
                                         func_r_d,
                                         Altitudeh_d,
                                         Altitude_d,
                                         sim.A,
                                         point_local_d,
                                         point_num,
                                         1,
                                         sim.DeepModel);
        }

        BENCH_POINT_I_S(current_step, rk, "DivM_Op_Poles", (), ("DivM_d", "divg_Mh_d"))

        //
        //      Slow Modes
        hipDeviceSynchronize();


        // Updates: SlowMh_d, SlowWh_d, SlowRho_d, Slowpressure_d
        Compute_Slow_Modes<LN, LN><<<NB, NT>>>(SlowMh_d,
                                               SlowWh_d,
                                               SlowRho_d,
                                               Slowpressure_d,
                                               Mhk_d,
                                               Whk_d,
                                               Rhok_d,
                                               Adv_d,
                                               DivM_d,
                                               diffmh_d,
                                               diffw_d,
                                               diffrh_d,
                                               diffpr_d,
                                               pressurek_d,
                                               h_d,
                                               hh_d,
                                               gtil_d,
                                               grad_d,
                                               div_d,
                                               Altitude_d,
                                               Altitudeh_d,
                                               sim.A,
                                               sim.Gravit,
                                               sim.Cp,
                                               sim.Rd,
                                               func_r_d,
                                               maps_d,
                                               nl_region,
                                               sim.DeepModel,
                                               sim.NonHydro);
        hipDeviceSynchronize();
        // Updates: SlowMh_d, SlowWh_d, SlowRho_d, Slowpressure_d
        Compute_Slow_Modes_Poles<6><<<2, 1>>>(SlowMh_d,
                                              SlowWh_d,
                                              SlowRho_d,
                                              Slowpressure_d,
                                              Mhk_d,
                                              Whk_d,
                                              Rhok_d,
                                              Adv_d,
                                              DivM_d,
                                              diffmh_d,
                                              diffw_d,
                                              diffrh_d,
                                              diffpr_d,
                                              pressurek_d,
                                              h_d,
                                              hh_d,
                                              gtil_d,
                                              grad_d,
                                              div_d,
                                              Altitude_d,
                                              Altitudeh_d,
                                              sim.A,
                                              sim.Gravit,
                                              sim.Cp,
                                              sim.Rd,
                                              func_r_d,
                                              point_local_d,
                                              nv,
                                              point_num,
                                              sim.DeepModel,
                                              sim.NonHydro);


        BENCH_POINT_I_S(current_step, rk, "Compute_Slow_Modes_Poles", (), ("SlowMh_d", "SlowWh_d", "SlowRho_d", "Slowpressure_d"))

        //      Updates or initializes deviations.
        if (rk > 0) {
            hipDeviceSynchronize();

            BENCH_POINT_I_S_PHY(current_step, rk, "bRK", (), ("Rhos_d", "Rhok_d", "Mhs_d", "Mhk_d", "Whs_d", "Whk_d", "pressures_d", "pressurek_d", "pressure_d"))

            // Updates: Mhs_d, Whs_d, Ws_d, Rhos_d, pressures_d
            UpdateRK<<<(point_num / NTH) + 1, NTH>>>(Mhs_d,
                                                     Mhk_d,
                                                     Mh_d,
                                                     Whs_d,
                                                     Whk_d,
                                                     Wh_d,
                                                     Ws_d,
                                                     Rhos_d,
                                                     Rhok_d,
                                                     Rho_d,
                                                     pressures_d,
                                                     pressurek_d,
                                                     pressure_d,
                                                     func_r_d,
                                                     Altitude_d,
                                                     Altitudeh_d,
                                                     point_num,
                                                     nv);

            BENCH_POINT_I_S_PHY(current_step, rk, "RK", (), ("Rhos_d", "Rhok_d", "Mhs_d", "Mhk_d", "Whs_d", "Whk_d", "pressures_d", "pressurek_d", "pressure_d"))
        }

        //
        //      SMALL-STEPS
        // printf("\n << nlarge = %d <<<<<<<<<<<<\n",rk);
        for (int ns = 0; ns < ns_it; ns++) {
            // printf("// nsmall = %d //////////////////\n",ns);
            hipMemset(DivM_d, 0, sizeof(double) * point_num * 3 * nv);
            hipMemset(divg_Mh_d, 0, sizeof(double) * point_num * 3 * nv);
            if (sim.DivDampP) {
                hipDeviceSynchronize();
                // Updates: DivM_d, divg_Mh_d
                DivM_Op<LN, LN><<<NB, NT>>>(DivM_d,
                                            divg_Mh_d,
                                            Mhs_d,
                                            Whs_d,
                                            Kdhz_d,
                                            areasTr_d,
                                            nvecoa_d,
                                            nvecti_d,
                                            nvecte_d,
                                            func_r_d,
                                            Altitudeh_d,
                                            Altitude_d,
                                            sim.A,
                                            maps_d,
                                            nl_region,
                                            0,
                                            sim.DeepModel);
                // Updates: DivM_d, divg_Mh_d
                DivM_Op_Poles<5><<<NBP, 1>>>(DivM_d,
                                             divg_Mh_d,
                                             Mhs_d,
                                             Whs_d,
                                             Kdhz_d,
                                             areasTr_d,
                                             nvecoa_d,
                                             nvecti_d,
                                             nvecte_d,
                                             func_r_d,
                                             Altitudeh_d,
                                             Altitude_d,
                                             sim.A,
                                             point_local_d,
                                             point_num,
                                             0,
                                             sim.DeepModel);

                hipDeviceSynchronize();
                // Updates: DivM_d, divg_Mh_d
                DivM_Op<LN, LN><<<NB, NT>>>(DivM_d,
                                            divg_Mh_d,
                                            Mhs_d,
                                            Whs_d,
                                            Kdhz_d,
                                            areasTr_d,
                                            nvecoa_d,
                                            nvecti_d,
                                            nvecte_d,
                                            func_r_d,
                                            Altitudeh_d,
                                            Altitude_d,
                                            sim.A,
                                            maps_d,
                                            nl_region,
                                            1,
                                            sim.DeepModel);
                // Updates: DivM_d, divg_Mh_d
                DivM_Op_Poles<5><<<NBP, 1>>>(DivM_d,
                                             divg_Mh_d,
                                             Mhs_d,
                                             Whs_d,
                                             Kdhz_d,
                                             areasTr_d,
                                             nvecoa_d,
                                             nvecti_d,
                                             nvecte_d,
                                             func_r_d,
                                             Altitudeh_d,
                                             Altitude_d,
                                             sim.A,
                                             point_local_d,
                                             point_num,
                                             1,
                                             sim.DeepModel);

                BENCH_POINT_I_SS(current_step, rk, ns, "DivM_Op_Poles", (), ("DivM_d", "divg_Mh_d"))
            }

            //          Momentum equation.
            hipDeviceSynchronize();
            // Updates: Mhs_d
            Momentum_Eq<LN, LN><<<NB, NT>>>(Mhs_d,
                                            pressures_d,
                                            SlowMh_d,
                                            grad_d,
                                            Altitude_d,
                                            diffmh_d,
                                            sim.A,
                                            func_r_d,
                                            times,
                                            maps_d,
                                            nl_region,
                                            sim.DeepModel);
            // Updates: Mhs_d
            Momentum_Eq_Poles<6><<<2, 1>>>(Mhs_d,
                                           pressures_d,
                                           SlowMh_d,
                                           grad_d,
                                           Altitude_d,
                                           DivM_d,
                                           sim.A,
                                           func_r_d,
                                           times,
                                           point_local_d,
                                           nv,
                                           point_num,
                                           sim.DeepModel);

            //          Vertical Momentum
            hipDeviceSynchronize();

            BENCH_POINT_I_SS(current_step, rk, ns, "Momentum_Eq", (), ("Rho_d", "pressures_d", "Mhs_d", "Wh_d", "temperature_d", "W_d"))
            // Updates: Sp_d, Sd_d
            Prepare_Implicit_Vertical<LN, LN><<<NB, NT>>>(Mhs_d,
                                                          h_d,
                                                          div_d,
                                                          Slowpressure_d,
                                                          SlowRho_d,
                                                          Sp_d,
                                                          Sd_d,
                                                          Altitude_d,
                                                          sim.Cp,
                                                          sim.Rd,
                                                          sim.A,
                                                          maps_d,
                                                          nl_region,
                                                          sim.DeepModel);


            hipDeviceSynchronize();
            // Updates: Sp_d, Sd_d
            Prepare_Implicit_Vertical_Poles<6><<<2, 1>>>(Mhs_d,
                                                         h_d,
                                                         div_d,
                                                         Slowpressure_d,
                                                         SlowRho_d,
                                                         Sp_d,
                                                         Sd_d,
                                                         Altitude_d,
                                                         sim.Cp,
                                                         sim.Rd,
                                                         sim.A,
                                                         point_local_d,
                                                         point_num,
                                                         nv,
                                                         sim.DeepModel);

            BENCH_POINT_I_SS(current_step, rk, ns, "Prepare_Implicit_Vertical", (), ("Sp_d", "Sd_d"))

            hipDeviceSynchronize();

            // Updates: Whs_d, Ws_d
            Vertical_Eq<<<(point_num / num_th_vertical_eq) + 1,
                          num_th_vertical_eq,
                          2 * num_th_vertical_eq * nvi * sizeof(double)>>>(Whs_d,
                                                                           Ws_d,
                                                                           pressures_d,
                                                                           h_d,
                                                                           hh_d,
                                                                           Rhos_d,
                                                                           gtil_d,
                                                                           gtilh_d,
                                                                           Sp_d,
                                                                           Sd_d,
                                                                           SlowWh_d,
                                                                           sim.Cp,
                                                                           sim.Rd,
                                                                           times,
                                                                           sim.Gravit,
                                                                           Altitude_d,
                                                                           Altitudeh_d,
                                                                           sim.A,
                                                                           sim.NonHydro,
                                                                           point_num,
                                                                           nv,
                                                                           nvi,
                                                                           sim.DeepModel);

            hipError_t err = hipGetLastError();


            // Check device query
            if (err != hipSuccess) {
                printf("thor_driver.cu: CUDA error check reports error: %s\n", hipGetErrorString(err));
            }

            //          Pressure and density equations.
            hipDeviceSynchronize();
            BENCH_POINT_I_SS_PHY(current_step, rk, ns, "Vertical_Eq", (), ("Whs_d", "Ws_d", "pressures_d", "h_d", "hh_d", "Rhos_d"));

            // update the physics modules in fast mode
            if (phy_modules_execute)
                phy_modules_dyn_core_loop_fast_modes(*this,
                                                     sim,
                                                     current_step,
                                                     times);


            BENCH_POINT_I_SS_PHY(current_step, rk, ns, "Phy_mod_fast_mode", (), ("Whs_d", "Ws_d", "pressures_d", "h_d", "hh_d", "Rhos_d"))

            // Updates: pressures_d, Rhos_d
            Density_Pressure_Eqs<LN, LN><<<NB, NT>>>(pressures_d,
                                                     pressurek_d,
                                                     Rhos_d,
                                                     Rhok_d,
                                                     Mhs_d,
                                                     Mhk_d,
                                                     Whs_d,
                                                     Whk_d,
                                                     pt_d,
                                                     pth_d,
                                                     SlowRho_d,
                                                     diffpr_d,
                                                     div_d,
                                                     Altitude_d,
                                                     Altitudeh_d,
                                                     sim.Cp,
                                                     sim.Rd,
                                                     sim.A,
                                                     sim.P_Ref,
                                                     times,
                                                     maps_d,
                                                     nl_region,
                                                     sim.DeepModel);

            hipDeviceSynchronize();
            // Updates: pressures_d, Rhos_d
            Density_Pressure_Eqs_Poles<6><<<2, 1>>>(pressures_d,
                                                    pressurek_d,
                                                    Rhos_d,
                                                    Rhok_d,
                                                    Mhs_d,
                                                    Mhk_d,
                                                    Whs_d,
                                                    Whk_d,
                                                    pt_d,
                                                    pth_d,
                                                    SlowRho_d,
                                                    diffpr_d,
                                                    div_d,
                                                    Altitude_d,
                                                    Altitudeh_d,
                                                    sim.Cp,
                                                    sim.Rd,
                                                    sim.A,
                                                    sim.P_Ref,
                                                    times,
                                                    point_local_d,
                                                    point_num,
                                                    nv,
                                                    sim.DeepModel);

            BENCH_POINT_I_SS(current_step, rk, ns, "Density_Pressure_Eqs", (), ("pressures_d", "Rhos_d"))
        }
        BENCH_POINT_I_S_PHY(current_step, rk, "bRK2", (), ("Rhos_d", "Rhok_d", "Mhs_d", "Mhk_d", "Whs_d", "Whk_d", "pressures_d", "pressurek_d"))
        //      Update quantities for the long loop.
        hipDeviceSynchronize();
        // Updates: Mhk_d, Whk_d, Wk_d, Rhok_d, pressurek_d
        UpdateRK2<<<(point_num / NTH) + 1, NTH>>>(Mhs_d,
                                                  Mhk_d,
                                                  Whs_d,
                                                  Whk_d,
                                                  Wk_d,
                                                  Rhos_d,
                                                  Rhok_d,
                                                  pressures_d,
                                                  pressurek_d,
                                                  func_r_d,
                                                  Altitude_d,
                                                  Altitudeh_d,
                                                  point_num,
                                                  nv);

        BENCH_POINT_I_S_PHY(current_step, rk, "RK2", (), ("Rhos_d", "Rhok_d", "Mhs_d", "Mhk_d", "Whs_d", "Whk_d", "pressures_d", "pressurek_d"))
    }
    //  Update diagnostic variables.
    hipDeviceSynchronize();

    BENCH_POINT_I_PHY(current_step, "END", (), ("Rho_d", "pressure_d", "Mh_d", "Wh_d", "temperature_d", "W_d"))

    hipMemcpy(Mh_d, Mhk_d, point_num * nv * 3 * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(Wh_d, Whk_d, point_num * nvi * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(W_d, Wk_d, point_num * nv * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(Rho_d, Rhok_d, point_num * nv * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(pressure_d, pressurek_d, point_num * nv * sizeof(double), hipMemcpyDeviceToDevice);

    if (phy_modules_execute)
        phy_modules_dyn_core_loop_end(*this);
}
//END OF THOR!
