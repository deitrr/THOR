#include "hip/hip_runtime.h"
// ==============================================================================
// This file is part of THOR.
//
//     THOR is free software : you can redistribute it and / or modify
//     it under the terms of the GNU General Public License as published by
//     the Free Software Foundation, either version 3 of the License, or
//     (at your option) any later version.
//
//     THOR is distributed in the hope that it will be useful,
//     but WITHOUT ANY WARRANTY; without even the implied warranty of
//     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
//     GNU General Public License for more details.
//
//     You find a copy of the GNU General Public License in the main
//     THOR directory under <license.txt>.If not, see
//     <http://www.gnu.org/licenses/>.
// ==============================================================================
//
// Build the class ESP (Exoclimes Simulation Platform)
//
//
// Description:
//   Declare and initialize variables in the model
//
// Method: -
//
//
// Known limitations: None.
//
//
// Known issues: None.
//
//
// If you use this code please cite the following reference:
//
//       [1] Mendonca, J.M., Grimm, S.L., Grosheintz, L., & Heng, K., ApJ, 829, 115, 2016
//
// Current Code Owner: Joao Mendonca, EEG. joao.mendonca@csh.unibe.ch
//
// History:
// Version Date       Comment
// ======= ====       =======
//
// 1.0     16/08/2017 Released version  (JM)
//
////////////////////////////////////////////////////////////////////////

#include "../headers/esp.h"
#include "hdf5.h"

__host__ ESP::ESP(int *point_local_    ,
                  int *maps_           ,
                  double *lonlat_      ,
                  double *Altitude_    ,
                  double *Altitudeh_   ,
                  double *nvecoa_      ,
                  double *nvecti_      ,
                  double *nvecte_      ,
                  double *areasT_      ,
                  double *areasTr_     ,
                  double *div_         ,
                  double *grad_        ,
                  double *func_r_      ,
                  int nl_region_       ,
                  int nr_              ,
                  int nv_              ,
                  int nvi_             ,
                  int point_num_    ): nl_region(nl_region_), nr(nr_), point_num(point_num_), nv(nv_), nvi(nvi_){

    point_local_h = point_local_;
    maps_h        = maps_       ;

    lonlat_h = lonlat_;

    Altitude_h = Altitude_ ;
    Altitudeh_h= Altitudeh_;

    nvecoa_h= nvecoa_    ;
    nvecti_h= nvecti_    ;
    nvecte_h= nvecte_    ;
    areasTr_h = areasTr_ ;
    areasT_h= areasT_    ;

    div_h = div_ ;
    grad_h= grad_;

    func_r_h = func_r_ ;

//
//  Allocate Data
    AllocData();
}

__host__ void ESP::AllocData(){


//
//  Description:
//
//  Allocate data on host and device.
//
//  Allocate data in host
//  Diagnostics
    Rho_h        = (double*)malloc(nv*point_num   * sizeof(double));
    pressure_h   = (double*)malloc(nv*point_num   * sizeof(double));
    temperature_h= (double*)malloc(nv*point_num   * sizeof(double));
    Mh_h         = (double*)malloc(nv*point_num*3 * sizeof(double));
    W_h          = (double*)malloc(nv*point_num   * sizeof(double));
    Wh_h         = (double*)malloc(nvi*point_num  * sizeof(double));

//  Allocate data in device
//  Grid
    hipMalloc((void **)&point_local_d, 6 * point_num * sizeof(int));
    hipMalloc((void **)&maps_d, (nl_region + 2)*(nl_region + 2)*nr * sizeof(int));

//  Operators
    hipMalloc((void **)&nvecoa_d , 6 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&nvecti_d , 6 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&nvecte_d , 6 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&areasTr_d, 6 * point_num * sizeof(double));
    hipMalloc((void **)&func_r_d  , 3 * point_num * sizeof(double));
    hipMalloc((void **)&div_d, 7 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&grad_d,7 * 3 * point_num * sizeof(double));

//  Altitude (grid)
    hipMalloc((void **)&Altitude_d  , nv   * sizeof(double));
    hipMalloc((void **)&Altitudeh_d , nvi  * sizeof(double));

//  Longitude-latitude
    hipMalloc((void **)&lonlat_d  , 2 * point_num * sizeof(double));

//  Diagnostics
    hipMalloc((void **)&Mh_d         , nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&W_d          , nv * point_num *     sizeof(double));
    hipMalloc((void **)&Wh_d         , nvi* point_num *     sizeof(double));
    hipMalloc((void **)&Rho_d        , nv * point_num   * sizeof(double));
    hipMalloc((void **)&pressure_d   , nv * point_num   * sizeof(double));

//  Temperature
    hipMalloc((void **)&temperature_d, nv * point_num *     sizeof(double));

//  Potential temperature
    hipMalloc((void **)&pt_d         , nv * point_num *     sizeof(double));
    hipMalloc((void **)&pth_d        , nvi* point_num *     sizeof(double));

//  Entalphy
    hipMalloc((void **)&h_d           , nv * point_num *     sizeof(double));
    hipMalloc((void **)&hh_d          , nvi * point_num *     sizeof(double));

//  Advection
    hipMalloc((void **)&Adv_d        , nv * point_num * 3 * sizeof(double));

//  3D vector
    hipMalloc((void **)&v_d          , nv * point_num * 3 * sizeof(double));

//  Effective gravity
    hipMalloc((void **)&gtil_d        , nv * point_num * sizeof(double));
    hipMalloc((void **)&gtilh_d       , nvi* point_num * sizeof(double));

//  Slow modes
    hipMalloc((void **)&SlowMh_d        , nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&SlowWh_d        , nvi* point_num *     sizeof(double));
    hipMalloc((void **)&SlowRho_d       , nv * point_num *     sizeof(double));
    hipMalloc((void **)&Slowpressure_d  , nv * point_num *     sizeof(double));


//  Deviations
    hipMalloc((void **)&pressures_d   , nv * point_num *     sizeof(double));
    hipMalloc((void **)&Rhos_d        , nv * point_num *     sizeof(double));
    hipMalloc((void **)&Mhs_d         , nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&Ws_d          , nv * point_num *     sizeof(double));
    hipMalloc((void **)&Whs_d         , nvi* point_num *     sizeof(double));



//  RK-Method
    hipMalloc((void **)&pressurek_d   , nv * point_num *     sizeof(double));
    hipMalloc((void **)&Rhok_d        , nv * point_num *     sizeof(double));
    hipMalloc((void **)&Mhk_d         , nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&Wk_d          , nv * point_num *     sizeof(double));
    hipMalloc((void **)&Whk_d         , nvi* point_num *     sizeof(double));

//  Vertical integration
    hipMalloc((void **)&Sp_d          , nv * point_num * sizeof(double));
    hipMalloc((void **)&Sd_d          , nv * point_num * sizeof(double));

//  Diffusion
    hipMalloc((void **)&Kdhz_d         ,nv *                 sizeof(double));
    hipMalloc((void **)&Kdh4_d         ,nv *                 sizeof(double));
    hipMalloc((void **)&DivM_d         ,nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&diffpr_d       ,nv * point_num     * sizeof(double));
    hipMalloc((void **)&diffmh_d       , 3 * nv * point_num* sizeof(double));
    hipMalloc((void **)&diffw_d        , nv* point_num     * sizeof(double));
    hipMalloc((void **)&diffrh_d       , nv * point_num    * sizeof(double));
    hipMalloc((void **)&diff_d          , 6 * nv * point_num    * sizeof(double));
    hipMalloc((void **)&divg_Mh_d       , 3 * nv * point_num    * sizeof(double));

//  Mixing ratios 
    hipMalloc((void **)&mixH_d          , nv * point_num *     sizeof(double));

//  Extras-nan
    hipMalloc((void **)&check_d, sizeof (bool));

}

__host__ void ESP::InitialValues(bool rest          ,
                                 int glevel         ,
                                 double timestep_dyn,
                                 double A           ,
                                 double Cp          ,
                                 double P_Ref       ,
                                 double Gravit      ,
                                 double Omega       ,
                                 double Diffc       ,
                                 double kb          ,
                                 double Tmean       ,
                                 double Mmol        ,
                                 double mu          ,
                                 double Rd          ){
//
//  Description:
//
//  Set initial conditions.
//
//
//  Initial atmospheric conditions
    if(rest){
        for (int i = 0; i < point_num; i++ ){
//
//          Initial conditions for an isothermal Atmosphere
//
            double Ha = Rd * Tmean / Gravit;
            for (int lev = 0; lev < nv; lev++ ){
                pressure_h[i*nv + lev] = P_Ref*exp(-Altitude_h[lev] / Ha);
                temperature_h[i*nv + lev] = Tmean;
            }

            for (int lev = 0; lev < nv; lev++ ){
//              Density [kg/m3]
                Rho_h[i*nv + lev] = pressure_h[i*nv + lev] / (temperature_h[i*nv + lev] * Rd);

//              Momentum [kg/m3 m/s]
                Mh_h[i*3*nv + 3*lev + 0] = 0.0;
                Mh_h[i*3*nv + 3*lev + 1] = 0.0;
                Mh_h[i*3*nv + 3*lev + 2] = 0.0;

//              Vertical momentum [kg/m3 m/s]
                W_h[i*nv + lev] = 0.0;     // Center of the layer.
                Wh_h[i*(nv+1) + lev] = 0.0;// Layers interface.
            }
            Wh_h[i*(nv + 1) + nv] = 0.0;
        }
    }
    else{
//
//      Restart from an existing simulation.
        hid_t       file_id, dataset_id;

        char FILE_NAME1[160];

        sprintf(FILE_NAME1, "ifile/esp_initial.h5");
        file_id = H5Fopen(FILE_NAME1, H5F_ACC_RDONLY, H5P_DEFAULT);
//      Density
        dataset_id = H5Dopen(file_id, "/Rho",H5P_DEFAULT);
        H5Dread(dataset_id, H5T_NATIVE_DOUBLE, H5S_ALL, H5S_ALL, H5P_DEFAULT, Rho_h);
//      Pressure
        dataset_id = H5Dopen(file_id, "/Pressure",H5P_DEFAULT);
        H5Dread(dataset_id, H5T_NATIVE_DOUBLE, H5S_ALL, H5S_ALL, H5P_DEFAULT, pressure_h);
//      Horizontal momentum
        dataset_id = H5Dopen(file_id, "/Mh",H5P_DEFAULT);
        H5Dread(dataset_id, H5T_NATIVE_DOUBLE, H5S_ALL, H5S_ALL, H5P_DEFAULT, Mh_h);
//      Vertical momentum
        dataset_id = H5Dopen(file_id, "/Wh",H5P_DEFAULT);
        H5Dread(dataset_id, H5T_NATIVE_DOUBLE, H5S_ALL, H5S_ALL, H5P_DEFAULT, Wh_h);

        H5Dclose(dataset_id);
        H5Fclose(file_id);

        for(int lev = 0; lev < nv+1; lev++)
            for(int i = 0; i < point_num; i++)
                temperature_h[i*nv + lev] = pressure_h[i*nv + lev]/(Rd*Rho_h[i*nv + lev]);

        for(int i = 0; i < point_num; i++){
            for(int lev = 1; lev < nv; lev++){
                double xi  = Altitude_h[lev  ] ;
                double xim1= Altitudeh_h[lev ] ;
                double xip1= Altitudeh_h[lev +1  ] ;

                double a = (xi - xip1)/(xim1 -xip1);
                double b = (xi - xim1)/(xip1 -xim1);

                W_h[i*nv + lev] = Wh_h[i*(nv+1) + lev]*a + Wh_h[i*(nv+1) + lev+1]*b;
            }
        }
    }


//  Diffusion
//  Horizontal
    double *Kdhz_h, *Kdh4_h;
    Kdhz_h = new double[nv];
    Kdh4_h = new double[nv];
    for (int lev = 0; lev < nv; lev++ ){
//      Diffusion constant.
        double dbar = sqrt(2*M_PI/5)*A/(pow(2,glevel));
        Kdh4_h[lev] = Diffc*pow(dbar,4.)/timestep_dyn;
        Kdhz_h[lev] = Diffc*pow(dbar,4.)/timestep_dyn;
    }

//  Copy memory to the devide
    hipMemcpy(point_local_d, point_local_h, 6 * point_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(maps_d, maps_h, (nl_region + 2)*(nl_region + 2)*nr * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Altitude_d , Altitude_h , nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Altitudeh_d, Altitudeh_h, nvi* sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nvecoa_d  , nvecoa_h  , 6 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nvecti_d  , nvecti_h  , 6 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nvecte_d  , nvecte_h  , 6 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(areasTr_d , areasTr_h , 6 *     point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(lonlat_d , lonlat_h , 2 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(func_r_d  , func_r_h  , 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temperature_d, temperature_h, point_num * nv    *     sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Mh_d         , Mh_h         , point_num * nv    * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(W_d          , W_h          , point_num * nv    *     sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Wh_d         , Wh_h         , point_num * nvi   *     sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Rho_d        , Rho_h         , point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(pressure_d   , pressure_h    , point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(div_d, div_h, 7 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(grad_d,grad_h,7 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Kdhz_d      ,Kdhz_h, nv     * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Kdh4_d      ,Kdh4_h, nv     * sizeof(double), hipMemcpyHostToDevice);

//  Initialize arrays
    hipMemset(Adv_d, 0, sizeof(double) * 3 * point_num * nv);
    hipMemset(v_d  , 0, sizeof(double) * nv * point_num * 3);
    hipMemset(pt_d , 0, sizeof(double) * nv * point_num    );
    hipMemset(pth_d, 0, sizeof(double) * nvi* point_num    );
    hipMemset(SlowMh_d        , 0, sizeof(double) * nv * point_num * 3);
    hipMemset(SlowWh_d        , 0, sizeof(double) * nvi* point_num    );
    hipMemset(SlowRho_d       , 0, sizeof(double) * nv * point_num    );
    hipMemset(Slowpressure_d  , 0, sizeof(double) * nv * point_num    );
    hipMemset(h_d        , 0, sizeof(double) * nv * point_num    );
    hipMemset(hh_d       , 0, sizeof(double) * nvi * point_num    );
    hipMemset(Rhos_d     , 0, sizeof(double) * nv * point_num    );
    hipMemset(pressures_d, 0, sizeof(double) * nv * point_num    );
    hipMemset(Mhs_d      , 0, sizeof(double) * nv * point_num * 3);
    hipMemset(Ws_d       , 0, sizeof(double) * nv * point_num    );
    hipMemset(Whs_d      , 0, sizeof(double) * nvi* point_num    );
    hipMemset(gtil_d   , 0, sizeof(double) * nv * point_num);
    hipMemset(gtilh_d  , 0, sizeof(double) * nvi* point_num);
    hipMemset(Rhok_d     , 0, sizeof(double) * nv * point_num    );
    hipMemset(pressurek_d, 0, sizeof(double) * nv * point_num    );
    hipMemset(Mhk_d      , 0, sizeof(double) * nv * point_num * 3);
    hipMemset(Wk_d       , 0, sizeof(double) * nv * point_num    );
    hipMemset(Whk_d      , 0, sizeof(double) * nvi* point_num    );
    hipMemset(Sp_d       , 0, sizeof(double) * point_num * nv);
    hipMemset(Sd_d       , 0, sizeof(double) * point_num * nv);
    hipMemset(DivM_d      , 0, sizeof(double) * point_num * 3 * nv);
    hipMemset(diffpr_d    , 0, sizeof(double) * nv * point_num);
    hipMemset(diffmh_d    , 0, sizeof(double) * 3 * nv * point_num);
    hipMemset(diffw_d     , 0, sizeof(double) * nv * point_num);
    hipMemset(diffrh_d    , 0, sizeof(double) * nv * point_num);
    hipMemset(diff_d       , 0, sizeof(double) * 6 * nv * point_num);
    hipMemset(divg_Mh_d    , 0, sizeof(double) * 3 * nv * point_num);

    delete [] Kdh4_h;
    delete [] Kdhz_h;
}

__host__ ESP::~ESP(){

//
//  Description: Frees the memory space.
//
//  Host
    free(point_local_h);
    free(maps_h);
    free(lonlat_h);
    free(Altitude_h);
    free(Altitudeh_h);
    free(nvecoa_h);
    free(nvecti_h);
    free(nvecte_h);
    free(areasTr_h);
    free(div_h);
    free(grad_h);
    free(func_r_h);
    free(Rho_h);
    free(pressure_h);
    free(temperature_h);
    free(Mh_h);
    free(W_h);
    free(Wh_h);

//  Device
    hipFree(point_local_d);
    hipFree(maps_d);
    hipFree(Altitude_d);
    hipFree(Altitudeh_d);
    hipFree(nvecoa_d);
    hipFree(nvecti_d);
    hipFree(nvecte_d);
    hipFree(areasTr_d);
    hipFree(lonlat_d);
    hipFree(div_d);
    hipFree(grad_d);
    hipFree(func_r_d);
    hipFree(Rho_d);
    hipFree(pressure_d);
    hipFree(temperature_d);
    hipFree(W_d);
    hipFree(Wh_d);
    hipFree(h_d);
    hipFree(hh_d);
    hipFree(Adv_d);
    hipFree(gtil_d);
    hipFree(gtilh_d);
    hipFree(v_d);
    hipFree(pt_d);
    hipFree(pth_d);
    hipFree(SlowMh_d);
    hipFree(SlowWh_d);
    hipFree(SlowRho_d);
    hipFree(Slowpressure_d);
    hipFree(Rhok_d);
    hipFree(pressurek_d);
    hipFree(Mhk_d);
    hipFree(Whk_d);
    hipFree(Wk_d);
    hipFree(Rhos_d);
    hipFree(pressures_d);
    hipFree(Mhs_d);
    hipFree(Whs_d);
    hipFree(Ws_d);
    hipFree(Sd_d);
    hipFree(Sp_d);
    hipFree(Kdhz_d);
    hipFree(Kdh4_d);
    hipFree(DivM_d);
    hipFree(diffpr_d);
    hipFree(diffmh_d);
    hipFree(diffw_d);
    hipFree(diffrh_d);
    hipFree(diff_d);
    hipFree(divg_Mh_d);

    printf("\n\n Free memory!\n\n");
}
